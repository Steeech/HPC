#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <ctime>
#include <stdio.h>
#include <cmath>
#define BLOCK_SIZE  16          
#define N           1024        

__global__ void matMult(float* a, float* b, int n, float* c)
{
    int bx = blockIdx.x;     
    int by = blockIdx.y;
    int tx = threadIdx.x;      
    int ty = threadIdx.y;
    float sum = 0.0f;           
    int ia = n * BLOCK_SIZE * by + n * ty;  
    int ib = BLOCK_SIZE * bx + tx;
    for (int k = 0; k < n; k++)
        sum += a[ia + k] * b[ib + k * n];
    int ic = n * BLOCK_SIZE * by + BLOCK_SIZE * bx;
    c[ic + n * ty + tx] = sum;
}

int main(int argc, char* argv[])
{
    int numBytes = N * N * sizeof(float);
    float* a = new float[N * N];
    float* b = new float[N * N];
    float* c = new float[N * N];
    srand(time(0));
    for (int i = 0; i < N; i++)
        for (int j = 0; j < N; j++)
        {
            int	k = N * i + j;

            a[k] = rand() % 10;
            b[k] = rand() % 10;
        }

    float* adev = NULL;
    float* bdev = NULL;
    float* cdev = NULL;

    hipMalloc((void**)&adev, numBytes);
    hipMalloc((void**)&bdev, numBytes);
    hipMalloc((void**)&cdev, numBytes);

    dim3 threads(BLOCK_SIZE, BLOCK_SIZE);
    dim3 blocks(N / threads.x, N / threads.y);

    hipEvent_t start, stop;
    float gpuTime = 0.0f;

    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start, 0);
    hipMemcpy(adev, a, numBytes, hipMemcpyHostToDevice);
    hipMemcpy(bdev, b, numBytes, hipMemcpyHostToDevice);

    matMult << <blocks, threads >> > (adev, bdev, N, cdev);

    hipMemcpy(c, cdev, numBytes, hipMemcpyDeviceToHost);
    hipEventRecord(stop, 0);

    hipEventSynchronize(stop);
    hipEventElapsedTime(&gpuTime, start, stop);

    printf("time spent executing by the GPU: %.2f millseconds\n", gpuTime);

   
    hipEventDestroy(start);
    hipEventDestroy(stop);
    hipFree(adev);
    hipFree(bdev);
    hipFree(cdev);

    float* c_cpu = new float[N * N];
    unsigned int cpu_start = clock();
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++)
        {
            float summ = 0;
            for (int k = 0; k < N; k++) {
                 summ+= a[i *N + k] * b[k*N + j];
            }
            c_cpu[i*N + j]=summ;
        }
    }
    unsigned int cpu_end = clock();
    double cpuTime = (double)(cpu_end - cpu_start);
    printf("time spent executing by the CPU: %.2f millseconds\n", cpuTime);

    delete a;
    delete b;
    delete c;
    return 0;
}
